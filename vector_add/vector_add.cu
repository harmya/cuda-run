#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define checkCudaErrors(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error: %s:%d, %s\n", __FILE__, __LINE__, \
                hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

__global__ void vectorAdd(float* a, float* b, float* c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

void initVector(float *vec, int n) {
    for (int i = 0; i < n; i++) {
        vec[i] = (float)rand() / RAND_MAX;
    }
}

void verifyResult(const float *A, const float *B, const float *C, int n) {
    for (int i = 0; i < n; i++) {
        float expected = A[i] + B[i];
        if (fabs(C[i] - expected) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            fprintf(stderr, "Expected: %f, Got: %f\n", expected, C[i]);
            exit(EXIT_FAILURE);
        }
    }
}

int main() {
    int n = 1000000;
    size_t size = n * sizeof(float);

    srand(time(NULL));

    float *h_A = (float*) malloc(size);
    float *h_B = (float*) malloc(size);
    float *h_C = (float*) malloc(size);

    initVector(h_A, n);
    initVector(h_B, n);

    float *d_A, *d_B, *d_C;
    checkCudaErrors(hipMalloc(&d_A, size));
    checkCudaErrors(hipMalloc(&d_B, size));
    checkCudaErrors(hipMalloc(&d_C, size));

    checkCudaErrors(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    checkCudaErrors(hipEventRecord(start, NULL));
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, n);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipEventRecord(stop, NULL));
    checkCudaErrors(hipEventSynchronize(stop));

    float milliseconds = 0;
    checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Execution time: %.4f milliseconds\n", milliseconds);

    checkCudaErrors(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));
    verifyResult(h_A, h_B, h_C, n);
    
    float gigaFlops = (n / milliseconds) * 1.0e-6; 
    printf("Performance: %.4f GFlop/s\n", gigaFlops);

    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));
    
    free(h_A);
    free(h_B);
    free(h_C);

    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    return 0;

}